#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
//    Copyright 2010
//
//    This file is part of rootpwa
//
//    rootpwa is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    rootpwa is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with rootpwa. If not, see <http://www.gnu.org/licenses/>.
//
///////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
//
// Description:
//      test processing of complex value arrays
//
//      template design based on ideas from
//      http://blog.icare3d.org/2010/06/cuda-dynamic-template-parameters-22.html
//
//
// Author List:
//      Boris Grube          TUM            (original author)
//
//
//-------------------------------------------------------------------------


#include <string>
#include <cstdlib>
#include <ctime>

#include <hip/hip_runtime_api.h>

#include "reportingUtils.hpp"
#include "arrayUtils.hpp"
#include "complexTest.cuh"
#include "textureReader.cuh"


using namespace std;
using namespace rpwa;


bool
printCudaDeviceInfo(const int deviceId)
{
        const unsigned int nGpuArchCoresPerSM[] = {1, 8, 32};  // from SDK/shared/inc/shrUtils.h

        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
        if (deviceId == 0) {
                // fields for both major & minor fields are 9999, if no CUDA capable devices are present
                if ((deviceProp.major == 9999) and (deviceProp.minor == 9999)) {
			printWarn << "there is no CUDA device" << endl;
			return false;
		}
	}
	printInfo << "CUDA device[" << deviceId << "]: '" << deviceProp.name << "'" << endl;
    
        // print info
        int driverVersion = 0;
        checkCudaErrors(hipDriverGetVersion(&driverVersion));
        int runtimeVersion = 0;     
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        cout << "    driver version: .................................. " << driverVersion / 1000 << "." << driverVersion % 100 << endl
             << "    runtime version: ................................. " << runtimeVersion / 1000 << "." << runtimeVersion % 100 << endl
             << "    capability major revision number: ................ " << deviceProp.major << endl
	     << "    capability minor revision number: ................ " << deviceProp.minor << endl
	     << "    GPU clock frequency: ............................. " << deviceProp.clockRate * 1e-6f << " GHz" << endl
	     << "    number of multiprocessors: ....................... " << deviceProp.multiProcessorCount << endl
	     << "    number of cores: ................................. " << nGpuArchCoresPerSM[deviceProp.major] * deviceProp.multiProcessorCount << endl
	     << "    warp size: ....................................... " << deviceProp.warpSize << endl
	     << "    maximum number of threads per block: ............. " << deviceProp.maxThreadsPerBlock << endl
	     << "    maximum block dimensions: ........................ " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1]
	     << " x " << deviceProp.maxThreadsDim[2] << endl
	     << "    maximum grid dimension ........................... " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1]
	     << " x " << deviceProp.maxGridSize[2] << endl
	     << "    total amount of global memory: ................... " << deviceProp.totalGlobalMem / (1024. * 1024. * 1024.) << " GiBytes" << endl
	     << "    total amount of constant memory: ................. " << deviceProp.totalConstMem << " bytes" << endl 
	     << "    total amount of shared memory per block: ......... " << deviceProp.sharedMemPerBlock << " bytes" << endl
	     << "    total number of registers available per block: ... " << deviceProp.regsPerBlock << endl
	     << "    maximum memory pitch: ............................ " << deviceProp.memPitch << " bytes" << endl
	     << "    texture alignment: ............................... " << deviceProp.textureAlignment << " bytes" << endl
	     << "    concurrent copy and execution: ................... " << ((deviceProp.deviceOverlap)            ? "yes" : "no") << endl
	     << "    run time limit on kernels: ....................... " << ((deviceProp.kernelExecTimeoutEnabled) ? "yes" : "no") << endl
	     << "    integrated: ...................................... " << ((deviceProp.integrated)               ? "yes" : "no") << endl
	     << "    support for host page-locked memory mapping: ..... " << ((deviceProp.canMapHostMemory)         ? "yes" : "no") << endl
	     << "    compute mode: .................................... " << ((deviceProp.computeMode == hipComputeModeDefault) ?
		"default (multiple host threads can use this device simultaneously)" :
		(deviceProp.computeMode == hipComputeModeExclusive) ?
		"exclusive (only one host thread at a time can use this device)" :
		(deviceProp.computeMode == hipComputeModeProhibited) ?
		"prohibited (no host thread can use this device)" :
		"unknown") << endl;
	return true;
}


template<typename T>
__global__
void
sumGlobalMemKernel(const T*           inData,   // pointer to device input data in global memory
                   T*                 outData,  // pointer to device output data in global memory
                   const unsigned int nmbElementsPerThread)
{
	const unsigned int threadId   = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int nmbThreads = gridDim.x * blockDim.x;
	T                  sum        = 0;
#pragma unroll 1
	for (unsigned int i = 0; i < nmbElementsPerThread; ++i)
		sum += inData[(i * nmbThreads) + threadId];  // coalesce memory access
	outData[threadId] = sum;
}


template<typename T, typename textureReaderT>
__global__
void
sumTextureMemKernel(T*                 outData,  // pointer to device output data in global memory
                    const unsigned int nmbElementsPerThread)
{
	const unsigned int threadId   = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int nmbThreads = gridDim.x * blockDim.x;
	T                  sum        = 0;
#pragma unroll 1
	for (unsigned int i = 0; i < nmbElementsPerThread; ++i)
		sum += textureReaderT::fetch((i * nmbThreads) + threadId);  // coalesce memory access
	outData[threadId] = sum;
}


template<typename T>
__global__
void
sum2GlobalMemKernel(const T*           inData,   // pointer to device input data in global memory
                    T*                 outData,  // pointer to device output data in global memory
                    const unsigned int nmbElements0,
                    const unsigned int nmbElements1)
{
	const unsigned int threadId   = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int nmbThreads = gridDim.x * blockDim.x;
	T                  sum        = 0;
	unsigned int       indices[2];
	const unsigned int dim[2] = {nmbElements0, nmbElements1};
#pragma unroll 1
	for (indices[0] = 0; indices[0] < dim[0]; ++indices[0])
		for (unsigned int i = 0; i < dim[1]; ++i) {
			indices[1] = (i * nmbThreads) + threadId;  // coalesce memory access
			sum += inData[indicesToOffset<unsigned int>(indices, dim, 2)];
		}
	outData[threadId] = sum;
}


template<typename T, typename textureReaderT>
__global__
void
sum2TextureMemKernel(T*                 outData,  // pointer to device output data in global memory
                     const unsigned int nmbElements0,
                     const unsigned int nmbElements1)
{
	const unsigned int threadId   = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int nmbThreads = gridDim.x * blockDim.x;
	T                  sum        = 0;
	unsigned int       indices[2];
	const unsigned int dim[2] = {nmbElements0, nmbElements1};
#pragma unroll 1
	for (indices[0] = 0; indices[0] < dim[0]; ++indices[0])
		for (unsigned int i = 0; i < dim[1]; ++i) {
			indices[1] = (i * nmbThreads) + threadId;  // coalesce memory access
			sum += textureReaderT::fetch(indicesToOffset<unsigned int>(indices, dim, 2));
		}
	outData[threadId] = sum;
}


template<typename T>
bool
verifySumKernel(const T*           inData,
                const T*           outData,  // output of GPU kernel
                const unsigned int nmbBlocks,
                const unsigned int nmbThreadsPerBlock,
                const unsigned int nmbElementsPerThread)
{
	if (not inData or not outData) {
		printWarn << "null pointer for data" << endl;
		return false;
	}
	const unsigned int nmbThreads = nmbBlocks * nmbThreadsPerBlock;
	T                  data[nmbThreads];
	bool               success = true;
	for (unsigned int threadId = 0; threadId < nmbThreads; ++threadId) {
		T sum = 0;
		for (unsigned int i = 0; i < nmbElementsPerThread; ++i)
			sum += inData[(i * nmbThreads ) + threadId];
		data[threadId] = sum;
		if (data[threadId] != outData[threadId]) {
			printWarn << "(CPU[" << threadId << "] = " << data[threadId]    << ") != "
			          << "(GPU[" << threadId << "] = " << outData[threadId] << "); "
			          << "delta = " << data[threadId] - outData[threadId] << endl;
			success = false;
		}
	}
	return success;
}


template<typename T>
bool
verifySum2Kernel(const T*            inData,
                 const T*            outData,  // output of GPU kernel
                 const unsigned int  nmbBlocks,
                 const unsigned int  nmbThreadsPerBlock,
                 const unsigned int* dim)
{
	if (not inData or not outData) {
		printWarn << "null pointer for data" << endl;
		return false;
	}
	const unsigned int nmbThreads = nmbBlocks * nmbThreadsPerBlock;
	T                  data[nmbThreads];
	bool               success = true;
	for (unsigned int threadId = 0; threadId < nmbThreads; ++threadId) {
		T            sum = 0;
		unsigned int indices[2];
		for (indices[0] = 0; indices[0] < dim[0]; ++indices[0])
			for (unsigned int i = 0; i < dim[1]; ++i) {
				indices[1] = (i * nmbThreads) + threadId;  // coalesce memory access
				sum += inData[indicesToOffset<unsigned int>(indices, dim, 2)];
			}
		data[threadId] = sum;
		if (data[threadId] != outData[threadId]) {
			printWarn << "(CPU[" << threadId << "] = " << data[threadId]    << ") != "
			          << "(GPU[" << threadId << "] = " << outData[threadId] << "); "
			          << "delta = " << data[threadId] - outData[threadId] << endl;
			success = false;
		}
	}
	return success;
}


template<typename T>
struct sumGlobalMemKernelCaller {
  
	typedef T value_type;

	static void call(const unsigned int nmbBlocks,
	                 const unsigned int nmbThreadsPerBlock,
	                 const T*           deviceInData,
	                 T*                 deviceOutData,
	                 const unsigned int nmbElementsPerThread)
	{
		sumGlobalMemKernel<T><<< nmbBlocks, nmbThreadsPerBlock >>>
			(deviceInData, deviceOutData, nmbElementsPerThread);
	}

	static unsigned long dataSize(const unsigned int nmbBlocks,
	                              const unsigned int nmbThreadsPerBlock,
	                              const unsigned int nmbElementsPerThread)
	{ return nmbBlocks * nmbThreadsPerBlock * nmbElementsPerThread * sizeof(T); }

	static bool verify(const T*           hostInData,
	                   const T*           hostOutData,  // output of GPU kernel
	                   const unsigned int nmbBlocks,
	                   const unsigned int nmbThreadsPerBlock,
	                   const unsigned int nmbElementsPerThread)
	{
		return verifySumKernel(hostInData, hostOutData, nmbBlocks,
		                       nmbThreadsPerBlock, nmbElementsPerThread);
	}

};


template<typename T, typename textureReaderT>
struct sumTextureMemKernelCaller {
  
	typedef T                                     value_type;
	typedef typename textureReaderT::texture_type texture_type;

	static void call(const unsigned int nmbBlocks,
	                 const unsigned int nmbThreadsPerBlock,
	                 const T*,
	                 T*                 deviceOutData,
	                 const unsigned int nmbElementsPerThread)
	{
		sumTextureMemKernel<T, textureReaderT><<< nmbBlocks, nmbThreadsPerBlock >>>
			(deviceOutData, nmbElementsPerThread);
	}

	static unsigned long dataSize(const unsigned int nmbBlocks,
	                              const unsigned int nmbThreadsPerBlock,
	                              const unsigned int nmbElementsPerThread)
	{ return nmbBlocks * nmbThreadsPerBlock * nmbElementsPerThread * sizeof(T); }

	static bool verify(const T*           hostInData,
	                   const T*           hostOutData,  // output of GPU kernel
	                   const unsigned int nmbBlocks,
	                   const unsigned int nmbThreadsPerBlock,
	                   const unsigned int nmbElementsPerThread)
	{
		return verifySumKernel(hostInData, hostOutData, nmbBlocks, nmbThreadsPerBlock,
		                       nmbElementsPerThread);
	}

};


template<typename T>
struct sum2GlobalMemKernelCaller {
  
	typedef T value_type;

	static void call(const unsigned int nmbBlocks,
	                 const unsigned int nmbThreadsPerBlock,
	                 const T*           deviceInData,
	                 T*                 deviceOutData,
	                 const unsigned int nmbElementsPerThread)
	{
		sum2GlobalMemKernel<T><<< nmbBlocks, nmbThreadsPerBlock >>>
			(deviceInData, deviceOutData, (unsigned int)sqrt(nmbElementsPerThread),
			 (unsigned int)sqrt(nmbElementsPerThread));
	}

	static unsigned long dataSize(const unsigned int nmbBlocks,
	                              const unsigned int nmbThreadsPerBlock,
	                              const unsigned int nmbElementsPerThread)
	{
		const unsigned int nmbElements[2] = {(unsigned int)sqrt(nmbElementsPerThread),
		                                     (unsigned int)sqrt(nmbElementsPerThread)};
		return nmbBlocks * nmbThreadsPerBlock * nmbElements[0] * nmbElements[1] * sizeof(T);
	}

	static bool verify(const T*           hostInData,
	                   const T*           hostOutData,  // output of GPU kernel
	                   const unsigned int nmbBlocks,
	                   const unsigned int nmbThreadsPerBlock,
	                   const unsigned int nmbElementsPerThread)
	{
		const unsigned int nmbElements[2] = {(unsigned int)sqrt(nmbElementsPerThread),
		                                     (unsigned int)sqrt(nmbElementsPerThread)};
		return verifySum2Kernel(hostInData, hostOutData, nmbBlocks, nmbThreadsPerBlock, nmbElements);
	}

};


template<typename T, typename textureReaderT>
struct sum2TextureMemKernelCaller {
  
	typedef T                                     value_type;
	typedef typename textureReaderT::texture_type texture_type;

	static void call(const unsigned int nmbBlocks,
	                 const unsigned int nmbThreadsPerBlock,
	                 const T*,
	                 T*                 deviceOutData,
	                 const unsigned int nmbElementsPerThread)
	{
		sum2TextureMemKernel<T, textureReaderT><<< nmbBlocks, nmbThreadsPerBlock >>>
			(deviceOutData, (unsigned int)sqrt(nmbElementsPerThread),
			 (unsigned int)sqrt(nmbElementsPerThread));
	}

	static unsigned long dataSize(const unsigned int nmbBlocks,
	                              const unsigned int nmbThreadsPerBlock,
	                              const unsigned int nmbElementsPerThread)
	{
		const unsigned int nmbElements[2] = {(unsigned int)sqrt(nmbElementsPerThread),
		                                     (unsigned int)sqrt(nmbElementsPerThread)};
		return nmbBlocks * nmbThreadsPerBlock * nmbElements[0] * nmbElements[1] * sizeof(T);
	}

	static bool verify(const T*           hostInData,
	                   const T*           hostOutData,  // output of GPU kernel
	                   const unsigned int nmbBlocks,
	                   const unsigned int nmbThreadsPerBlock,
	                   const unsigned int nmbElementsPerThread)
	{
		const unsigned int nmbElements[2] = {(unsigned int)sqrt(nmbElementsPerThread),
		                                     (unsigned int)sqrt(nmbElementsPerThread)};
		return verifySum2Kernel(hostInData, hostOutData, nmbBlocks, nmbThreadsPerBlock, nmbElements);
	}

};


template<typename T, typename textureReaderT, typename kernelCallerT>
void runKernel(const unsigned int nmbBlocks,
               const unsigned int nmbThreadsPerBlock,
               const unsigned int nmbIterations)
{
	// create maximum sized texture (512 MB)
	unsigned int       nmbElements          = ((unsigned int)1 << 29) / sizeof(T);
	const unsigned int nmbThreads           = nmbBlocks * nmbThreadsPerBlock;
	const unsigned int nmbElementsPerThread = nmbElements / nmbThreads;
	nmbElements = nmbElementsPerThread * nmbThreads;
	const unsigned int dataSizeIn  = nmbElements * sizeof(T);
	const unsigned int dataSizeOut = nmbThreads  * sizeof(T);

	// create and initalize host arrays
	printInfo << "allocating " << dataSizeIn / (1024. * 1024.) << " MiBytes in global memory "
	          << "(" << nmbElementsPerThread << " data elements per thread)" << endl;
	T* hostInData  = (T*) malloc(dataSizeIn );
	T* hostOutData = (T*) malloc(dataSizeOut);
	srand (time(NULL));
	for (unsigned int i = 0; i < nmbElements; ++i)
		hostInData[i] = (T)rand();

        // create device arrays and copy host data to device
        T* deviceInData;
        T* deviceOutData;
        checkCudaErrors(hipMalloc((void**) &deviceInData,  dataSizeIn ));
        checkCudaErrors(hipMalloc((void**) &deviceOutData, dataSizeOut));
        checkCudaErrors(hipMemcpy(deviceInData, hostInData, dataSizeIn, hipMemcpyHostToDevice));

        // bind texture
        textureReaderT::bindTexture(deviceInData, dataSizeIn);

	// dry-run kernel first to avoid any setup and caching effects
	kernelCallerT::call(nmbBlocks, nmbThreadsPerBlock, deviceInData,
	                    deviceOutData, nmbElementsPerThread);

        // setup and start timer
        hipEvent_t start, end;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&end  ));
        checkCudaErrors(hipEventRecord(start, 0));

        // run kernel
        for (unsigned int iteration = 0; iteration < nmbIterations; ++iteration)
		kernelCallerT::call(nmbBlocks, nmbThreadsPerBlock, deviceInData,
                                    deviceOutData, nmbElementsPerThread);

        // stop timer
        checkCudaErrors(hipEventRecord(end, 0));
        checkCudaErrors(hipEventSynchronize(end));

        // calculate and report bandwidth
        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
        elapsedTime /= nmbIterations * 1000;  // [sec] per iteration
        const unsigned long dataSize  = kernelCallerT::dataSize(nmbBlocks, nmbThreadsPerBlock,
                                                                nmbElementsPerThread);
	const float         bandwidth = dataSize / elapsedTime;
        printInfo << "processed " << dataSize / (1024. * 1024.) << " MiBytes in "
                  << elapsedTime * 1000 << " msec; "
                  << "total throughput: " << bandwidth / (1024 * 1024 * 1024) << " GiByte/sec" << endl;
        checkCudaErrors(hipEventDestroy(start));
        checkCudaErrors(hipEventDestroy(end  ));

        // copy kernel output data to host
        checkCudaErrors(hipMemcpy(hostOutData, deviceOutData, dataSizeOut, hipMemcpyDeviceToHost));

        // test data
        if (kernelCallerT::verify(hostInData, hostOutData, nmbBlocks,
	                          nmbThreadsPerBlock, nmbElementsPerThread))
		printInfo << "verification successful" << endl;
	else
		printWarn << "verification failed" << endl;

	// unbind texture
	textureReaderT::unbindTexture();

        // cleanup memory
        free(hostInData );
        free(hostOutData);
        checkCudaErrors(hipFree(deviceInData ));
        checkCudaErrors(hipFree(deviceOutData));
        hipDeviceReset();
}


int main(int,
         char**) 
{
        // get number of CUDA devices in system
        int deviceCount = 0;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
                printWarn << "there is no CUDA device" << endl;
                return 0;
	}
	printInfo << "found " << deviceCount << " CUDA device(s)" << endl;

	// print info for all CUDA devices in system
	for (int deviceId = 0; deviceId < deviceCount; ++deviceId)
                printCudaDeviceInfo(deviceId);
  
        // use most powerful GPU in system
        const int deviceId = gpuGetMaxGflopsDeviceId();
        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
        printInfo << "using CUDA device[" << deviceId << "]: '" << deviceProp.name << "'" << endl;
        checkCudaErrors(hipSetDevice(deviceId));

        // create maximum number of threads for all blocks
        const unsigned int nmbBlocks          = deviceProp.multiProcessorCount;
	const unsigned int nmbThreadsPerBlock = deviceProp.maxThreadsPerBlock;
	const unsigned int nmbIterations      = 100;
	printInfo << "using grid (" << nmbBlocks << " blocks) x "
	          << "(" << nmbThreadsPerBlock << " threads per block); "
	          << "running " << nmbIterations << " kernel iterations" << endl;
  
	// run kernels
	printInfo << "testing 1D complexTest<float> global memory read ---------------------------" << endl;
	runKernel<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader,
		sumGlobalMemKernelCaller<cuda::complexTest<float2, float> > >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);
	
	printInfo << "testing 1D complexTest<double> global memory read --------------------------" << endl;
	runKernel<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader,
		sumGlobalMemKernelCaller<cuda::complexTest<double2, double> > >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 1D complexTest<float> texture memory read --------------------------" << endl;
	runKernel<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader,
		sumTextureMemKernelCaller<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader> >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 1D complexTest<double> texture memory read -------------------------" << endl;
	runKernel<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader,
		sumTextureMemKernelCaller<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader> >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 2D complexTest<float> global memory read ---------------------------" << endl;
	runKernel<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader,
		sum2GlobalMemKernelCaller<cuda::complexTest<float2, float> > >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 2D complexTest<double> global memory read --------------------------" << endl;
	runKernel<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader,
		sum2GlobalMemKernelCaller<cuda::complexTest<double2, double> > >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 2D complexTest<float> texture memory read --------------------------" << endl;
	runKernel<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader,
		sum2TextureMemKernelCaller<cuda::complexTest<float2, float>, cuda::floatComplexTextureReader> >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);

	printInfo << "testing 2D complexTest<double> texture memory read -------------------------" << endl;
	runKernel<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader,
		sum2TextureMemKernelCaller<cuda::complexTest<double2, double>, cuda::doubleComplexTextureReader> >
		(nmbBlocks, nmbThreadsPerBlock, nmbIterations);
	
	return 0;
}
